#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <iostream>
using namespace std;

#define BLOCK_NUM 1                                                           
#define THREAD_NUM 1

/*./run
void updateUserSchedule2(float **trainMatrixdo, float **W, float **U, int **trainMatrixin, float **V, float **SV, float *Wi);


__global__ void updateUserCuda(float **prediction_items, float **rating_items, float **w_items, int userCount, int factors, float reg, float **w_cu1, float **u_cu, float **v_cu, float *wi_cu, float **sv_cu, float **train_spvdo_cu, int *train_n_cu, int **train_spvin_cu, float **v_col){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size_item;                                                              
  float ifv, ufget, tmp_uget;                                                 
  int i;                                                                      
  float res;                                                                  
  float numer, denom;                                                                                                                     
  //printf("1");
  //printf("%f ", prediction_items[4][0]);
  for(int u = bid*THREAD_NUM+tid; u < userCount; u+=BLOCK_NUM*THREAD_NUM){                                                                                    
   // printf("1");
    size_item = train_n_cu[u];                                                
    if (size_item == 0)        continue ;                                   
    for (int j = 0; j < size_item; j++) {                                     
      //printf("%d", j);
      i = train_spvin_cu[u][j];                                                        
      res = 0;                                                                
      for(int k=0; k<factors; k++){                                           
        res += u_cu[u][k] * v_cu[i][k];                         
      }                                                                       
     // printf("%f", res);
      //printf("%f", prediction_items[1][j]);
      prediction_items[u][j] = res;
      //printf("%f ", res);
      rating_items[u][j] = train_spvdo_cu[u][j];                                 
      w_items[u][j] = w_cu1[u][j];
      //printf("1");
    }                              
    //printf("1");
    for (int f = 0; f < factors; f++) {                                       
      numer = 0, denom = 0;                                                   
      for(int j = 0; j<size_item; j++){                                       
        i = train_spvin_cu[u][j];                                                      
        v_col[u][j] = v_cu[i][f];                                         
      }                                                                     
      for(int k = 0; k<factors; k++){                                         
        numer -= u_cu[u][k] * sv_cu[f][k];                        
      }                                                                         
     // printf("%f ", numer);
      ufget = u_cu[u][f];                                                
      for (int j = 0; j<size_item; j++) {                                       
        i = train_spvin_cu[u][j];                                                        
        ifv = v_col[u][j];                                                       
        prediction_items[u][j] -= ufget * ifv;                                     
        numer += (w_items[u][j] * rating_items[u][j] - (w_items[u][j] - wi_cu[i]) * prediction_items[u][j]) * ifv;
        denom += (w_items[u][j] - wi_cu[i]) * ifv * ifv;                           
      }                                                                       
      denom +=sv_cu[f][f] + reg;                                         
      u_cu[u][f] = numer / denom;                                        
      tmp_uget = numer / denom; 
      //printf("%f ", tmp_uget);
      //printf("1");
      for (int j = 0; j<size_item; j++){                                        
        prediction_items[u][j] += tmp_uget * v_col[u][j];                             
      }                                                                        
    }                                                          
  }                                                                           
}                 

__global__ void updateUserCuda2(float **prediction_items, float **rating_items, float **w_items, int userCount, int factors, float reg, float **w_cu1, float **u_cu, float **v_cu, float *wi_cu, float **sv_cu, float **train_spvdo_cu, int *train_n_cu, int **train_spvin_cu, float **v_col){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size_item;
  float ifv, ufget, tmp_uget;
  int i;
  float res;
  float numer, denom;

  int id =  bid*THREAD_NUM+tid;

  int divide = userCount/(BLOCK_NUM*THREAD_NUM);
  int last = userCount%(BLOCK_NUM*THREAD_NUM);
  int dis = divide;
  int u=0;
  if(id<last){

    dis++;
    u = (id-1)*dis;
    for(int i = 0; i < dis; i++){

      size_item = train_n_cu[u];
      if (size_item == 0)        continue ;
      for (int j = 0; j < size_item; j++) {
        i = train_spvin_cu[u][j];
        res = 0;
        for(int k=0; k<factors; k++){
          res += u_cu[u][k] * v_cu[i][k];
        }
        prediction_items[u][j] = res;
        rating_items[u][j] = train_spvdo_cu[u][j];
        w_items[u][j] = w_cu1[u][j];
      }

      for (int f = 0; f < factors; f++) {
        numer = 0, denom = 0;
        for(int j = 0; j<size_item; j++){
          i = train_spvin_cu[u][j];
          v_col[u][j] = v_cu[i][f];
        }
        for(int k = 0; k<factors; k++){
          numer -= u_cu[u][k] * sv_cu[f][k];
        }
        ufget = u_cu[u][f];
        for (int j = 0; j<size_item; j++) {
          i = train_spvin_cu[u][j];
          ifv = v_col[u][j];

          prediction_items[u][j] -= ufget * ifv;
          numer += (w_items[u][j] * rating_items[u][j] - (w_items[u][j] - wi_cu[i]) * prediction_items[u][j]) * ifv;
          denom += (w_items[u][j] - wi_cu[i]) * ifv * ifv;

        }
        denom +=sv_cu[f][f] + reg;
        u_cu[u][f] = numer / denom;
        tmp_uget = numer / denom;
        //printf("%f ", tmp_uget);
        for (int j = 0; j<size_item; j++){
          prediction_items[u][j] += tmp_uget * v_col[u][j];
        }
      }
      u++;
    }
  }
  else{
    u = last * (dis+1) + (id-last)*dis;
    for(int i = 0; i < dis; i++){
      size_item = train_n_cu[u];
      if (size_item == 0)        continue ;
      for (int j = 0; j < size_item; j++) {
        i = train_spvin_cu[u][j];
        res = 0;
        for(int k=0; k<factors; k++){
          res += u_cu[u][k] * v_cu[i][k];
        }
        prediction_items[u][j] = res;
        rating_items[u][j] = train_spvdo_cu[u][j];
        w_items[u][j] = w_cu1[u][j];
      }

      for (int f = 0; f < factors; f++) {
        numer = 0, denom = 0;
        for(int j = 0; j<size_item; j++){
          i = train_spvin_cu[u][j];
          v_col[u][j] = v_cu[i][f];
        }
        for(int k = 0; k<factors; k++){
          numer -= u_cu[u][k] * sv_cu[f][k];
        }
        ufget = u_cu[u][f];
        for (int j = 0; j<size_item; j++) {
          i = train_spvin_cu[u][j];
          ifv = v_col[u][j];

          prediction_items[u][j] -= ufget * ifv;
          numer += (w_items[u][j] * rating_items[u][j] - (w_items[u][j] - wi_cu[i]) * prediction_items[u][j]) * ifv;
          denom += (w_items[u][j] - wi_cu[i]) * ifv * ifv;

        }
        denom +=sv_cu[f][f] + reg;
        u_cu[u][f] = numer / denom;
        tmp_uget = numer / denom;
        //printf("%f ", tmp_uget);
        for (int j = 0; j<size_item; j++){
          prediction_items[u][j] += tmp_uget * v_col[u][j];
        }
      }
    }
  }

__global__ void updateUserCuda(float *prediction_items, float *rating_items, float *w_items, float *v_col, int uborder, int vborder, int userCount, int factors, float reg, float *w_cu, float *uvsv_cu, float *wi_cu, float *train_spvdo_cu, int *train_n_cu, int *train_spvin_cu){
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int size_item;
  float ifv, ufget, tmp_uget;
  int i;
  float res;
  float numer, denom;
  int size2;
  int index;
  for(int u = bid*THREAD_NUM+tid; u < userCount; u+=BLOCK_NUM*THREAD_NUM){
    size_item = train_n_cu[u+1] - train_n_cu[u];
    size2 = train_n_cu[u];
    if (size_item == 0)        continue ;
    for (int j = 0; j < size_item; j++) {
      index = size2+j;
      i = train_spvin_cu[index];
      res = 0;
      for(int k=0; k<factors; k++){
        res += uvsv_cu[u*factors+k] * uvsv_cu[uborder+i*factors+k];
      }
      prediction_items[index] = res;
      rating_items[index] = train_spvdo_cu[index];
      w_items[index] = w_cu[index];
    }
    for (int f = 0; f < factors; f++) {
      numer = 0, denom = 0;
      for(int j = 0; j<size_item; j++){
        i = train_spvin_cu[size2+j];
        v_col[size2+j] = uvsv_cu[uborder+i*factors+f];
      }
      for(int k = 0; k<factors; k++){
        numer -= uvsv_cu[u*factors+k] * uvsv_cu[vborder+f*factors+k];
      }
      ufget = uvsv_cu[u*factors+f];
      for (int j = 0; j<size_item; j++) {
        index = size2+j;
        i = train_spvin_cu[index];
        ifv = v_col[index];
        prediction_items[index] -= ufget * ifv;
        numer += (w_items[index] * rating_items[index] - (w_items[index] - wi_cu[i]) * prediction_items[index]) * ifv;
        denom += (w_items[index] - wi_cu[i]) * ifv * ifv;
      }
      denom += uvsv_cu[vborder+f*factors+f] + reg;
      uvsv_cu[u*factors+f] = numer / denom;
      tmp_uget = numer / denom;
      printf("%f ", tmp_uget);
      for (int j = 0; j<size_item; j++){
        prediction_items[size2+j] += tmp_uget * v_col[size2+j];
      }
    }
  }
//=======
//>>>>>>> da1f5daa4a28ef05be26223dd8ce61c51ef97c94
}
*/
/*
void updateUserSchedule2(float **trainMatrixdo, float **W, float **U, int **trainMatrixin, float **V, float **SV, float *Wi){
  int userCount = 10;                                                           
  int itemCount = 20;                                                           
  int factors = 8;                                                              
  float reg = 0;                                                                
  int max_size = 5, size, size2;
  float *w_items, *prediction_items, *v_col, *rating_items;
  int total_size = max_size * userCount + 10;
  hipMalloc((void**)&prediction_items,sizeof(float)*total_size);
  hipMalloc((void**)&w_items,sizeof(float)*total_size);
  hipMalloc((void**)&rating_items,sizeof(float)*total_size);
  hipMalloc((void**)&v_col,sizeof(float)*total_size);

  float *w_cu, *uvsv_cu, *train_spvdo_cu;
  int *train_spvin_cu;
  int uvsvSize = sizeof(float) * (userCount + itemCount + factors) * factors;
  hipMalloc((void**)&w_cu, sizeof(float)*total_size);
  hipMalloc((void**)&uvsv_cu, uvsvSize);
  hipMalloc((void**)&train_spvin_cu,sizeof(int)*total_size);
  hipMalloc((void**)&train_spvdo_cu,sizeof(float)*total_size);
  int *train_n, *train_n_cu;
  train_n = (int *)malloc(sizeof(int)*userCount);
  hipMalloc((void**)&train_n_cu,sizeof(int)*userCount);
  
  float *w_h, *uvsv_h, *train_spvdo_h;
  int *train_spvin_h;
  w_h = (float *)malloc(sizeof(float)*total_size);
  uvsv_h = (float *)malloc(uvsvSize);
  train_spvin_h = (int *)malloc(sizeof(int)*total_size);
  train_spvdo_h = (float *)malloc(sizeof(float)*total_size);
  train_n[0] = 0;
  //cout<<total_size<<endl;
  for (int u = 0; u < userCount; u++){
    size = 5;
    size2 = train_n[u];
    train_n[u+1] = size2 + size;
    for(int i=0; i<size; i++){
      //cout<<size2+i<<endl;
      w_h[size2+i] = W[u][i];
      train_spvin_h[size2+i] = trainMatrixin[u][i];
      train_spvdo_h[size2+i] = trainMatrixdo[u][i];
    }
  }
  int uborder = userCount * factors;
  int vborder = uborder + itemCount * factors;
  for (int u = 0; u < userCount; u++){
    for(int i=0; i<factors; i++)
      uvsv_h[u*factors+i] = U[u][i];
  } 
  for (int u = 0; u < itemCount; u++){
    for(int i=0; i<factors; i++)
      uvsv_h[uborder+u*factors+i] = V[u][i];
  } 
  for (int u = 0; u < factors; u++){
    for(int i=0; i<factors; i++)
      uvsv_h[vborder+u*factors+i] = SV[u][i];
  } 
  hipMemcpy(uvsv_cu, uvsv_h, uvsvSize, hipMemcpyHostToDevice);
  hipMemcpy(w_cu, w_h, sizeof(float)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_spvdo_cu, train_spvdo_h, sizeof(float)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_spvin_cu, train_spvin_h, sizeof(int)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_n_cu, train_n, sizeof(int)*userCount, hipMemcpyHostToDevice);
  float *wi_cu;
  hipMalloc((void**)&wi_cu, sizeof(float)*itemCount);
  hipMemcpy(wi_cu, Wi, sizeof(float)*itemCount, hipMemcpyHostToDevice);
  updateUserCuda<<<BLOCK_NUM,THREAD_NUM>>>(prediction_items, rating_items, w_items, v_col, uborder, vborder, userCount, factors, reg, w_cu, uvsv_cu, wi_cu, train_spvdo_cu, train_n_cu, train_spvin_cu);
  hipMemcpy(uvsv_h, uvsv_cu, uvsvSize, hipMemcpyDeviceToHost);
  for (int u = 0; u < userCount; u++){
    for(int i=0; i<factors; i++)
      U[u][i] = uvsv_h[u*factors+i];
  }
  cout<<endl;
  hipFree(prediction_items);
  hipFree(w_items);
  hipFree(rating_items);
  hipFree(v_col);
  //cout<<"470";
  hipFree(train_n_cu);
  hipFree(w_cu);
  hipFree(uvsv_cu);
  //cout<<"474";
  hipFree(train_spvin_cu);
  //cout<<"476";
  //hipFree(train_spvdo_cu);
  //cout<<"478";
  hipFree(wi_cu);
  //cout<<"480";
  free(train_n);
  free(w_h);
  free(uvsv_h);
  free(train_spvin_h);
  free(train_spvdo_h);
  hipFree(train_spvdo_cu);
  //cout<<"481";
}
/*
void updateUserSchedule2(float **trainMatrixdo, float **W, float **U, int **trainMatrixin, float **V, float **SV, float *Wi){                                          
  int max_size = 5, size;                                                   
  float **prediction_items;                                                      
  float **rating_items;                                                          
  float **w_items, **v_col;                                                       
                                                            
  hipMalloc((void**)&prediction_items,sizeof(float *)*userCount);                 
  hipMalloc((void**)&rating_items,sizeof(float *)*userCount);                     
  hipMalloc((void**)&w_items,sizeof(float *)*userCount);                          
  hipMalloc((void**)&v_col,sizeof(float *)*userCount);

  float **t1, **t2, **t3, **t4;
  t1 = (float **)malloc(sizeof(float *)*userCount);
  t2 = (float **)malloc(sizeof(float *)*userCount);  
  t3 = (float **)malloc(sizeof(float *)*userCount);  
  t4 = (float **)malloc(sizeof(float *)*userCount);  
  float *mm;
  mm = (float *)malloc(sizeof(float)*max_size);
  for(int i=0; i<max_size; i++)
    mm[i] = 0;
  for(int u=0; u<userCount; u++){
    float *tmp1, *tmp2, *tmp3, *tmp4;
    
    hipMalloc((void**)&tmp1,sizeof(float)*max_size);
    hipMalloc((void**)&tmp2,sizeof(float)*max_size);
    hipMalloc((void**)&tmp3,sizeof(float)*max_size);
    hipMalloc((void**)&tmp4,sizeof(float)*max_size);
    hipMemcpy(tmp1, mm, sizeof(float)*max_size, hipMemcpyHostToDevice);
    hipMemcpy(tmp2, mm, sizeof(float)*max_size, hipMemcpyHostToDevice);
    hipMemcpy(tmp3, mm, sizeof(float)*max_size, hipMemcpyHostToDevice);
    hipMemcpy(tmp4, mm, sizeof(float)*max_size, hipMemcpyHostToDevice);
    t1[u] = tmp1;
    t2[u] = tmp2;
    t3[u] = tmp3;
    t4[u] = tmp4;
  }
  hipMemcpy(prediction_items, t1, sizeof(float *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(rating_items, t2, sizeof(float *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(w_items, t3, sizeof(float *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(v_col, t4, sizeof(float *)*userCount, hipMemcpyHostToDevice);
                                      
  free(mm);
  float **w_cu, **u_cu, **v_cu, **sv_cu, **train_spvdo_cu;                      
  int **train_spvin_cu;                                                         
  hipMalloc((void**)&w_cu,sizeof(float *)*userCount);                          
  hipMalloc((void**)&u_cu,sizeof(float *)*userCount);                          
  hipMalloc((void**)&v_cu,sizeof(float *)*itemCount);                          
  hipMalloc((void**)&sv_cu,sizeof(float *)*factors);                           
  hipMalloc((void**)&train_spvin_cu,sizeof(int *)*userCount);                  
  hipMalloc((void**)&train_spvdo_cu,sizeof(float *)*userCount);                                                               
  int *train_n, *train_n_cu;                                                    
  train_n = (int *)malloc(sizeof(int)*userCount);                               
  hipMalloc((void**)&train_n_cu,sizeof(int)*userCount);                        
  
  float **w_h, **u_h, **v_h, **sv_h, **train_spvdo_h;                           
  int **train_spvin_h;                                                          
  w_h = (float **)malloc(sizeof(float *)*userCount);                            
  u_h = (float **)malloc(sizeof(float *)*userCount);                            
  v_h = (float **)malloc(sizeof(float *)*itemCount);                            
  sv_h = (float **)malloc(sizeof(float *)*factors);                             
  train_spvin_h = (int **)malloc(sizeof(int *)*userCount);                      
  train_spvdo_h = (float **)malloc(sizeof(float *)*userCount);                                                                  
  for (int u = 0; u < userCount; u++){                                          
    size = 5;                                               
    train_n[u] = size;                                                          
    float *tmp_train, *tmp_w, *tmp_u;                                           
    int *tmp;                                                                   
    hipMalloc((void**)&tmp_train,sizeof(float)*size);                          
    hipMalloc((void**)&tmp_w,sizeof(float)*size);                              
    hipMalloc((void**)&tmp_u,sizeof(float)*factors);                           
    hipMalloc((void**)&tmp,sizeof(int)*size);                 
    hipMemcpy(tmp_train, trainMatrixdo[u], sizeof(float)*size, hipMemcpyHostToDevice);
    hipMemcpy(tmp_w, W[u], sizeof(float)*size, hipMemcpyHostToDevice);
    hipMemcpy(tmp_u, U[u], sizeof(float)*factors, hipMemcpyHostToDevice);
    hipMemcpy(tmp, trainMatrixin[u], sizeof(int)*size, hipMemcpyHostToDevice);
    
    w_h[u] = tmp_w;                                                             
    u_h[u] = tmp_u;                                                             
    train_spvin_h[u] = tmp;                                                     
    train_spvdo_h[u] = tmp_train;                                               
  }                                                                             
                                                
  for (int u = 0; u < itemCount; u++){                                          
    float *tmp_v;                                                               
    hipMalloc((void**)&tmp_v,sizeof(float)*factors);                           
    hipMemcpy(tmp_v, V[u], sizeof(float)*factors, hipMemcpyHostToDevice);
    v_h[u] = tmp_v;                                                             
  }                                                                             
  for (int u = 0; u < factors; u++){                                            
    float *tmp_sv;                                                              
    hipMalloc((void**)&tmp_sv,sizeof(float)*factors);                          
    hipMemcpy(tmp_sv, SV[u], sizeof(float)*factors, hipMemcpyHostToDevice);
    sv_h[u] = tmp_sv;                                                           
  }                                                                             
                                                   
  hipMemcpy(u_cu, u_h, sizeof(float *)*userCount, hipMemcpyHostToDevice);     
  hipMemcpy(v_cu, v_h, sizeof(float *)*itemCount, hipMemcpyHostToDevice);     
  hipMemcpy(sv_cu, sv_h, sizeof(float *)*factors, hipMemcpyHostToDevice);     
  hipMemcpy(w_cu, w_h, sizeof(float *)*userCount, hipMemcpyHostToDevice);     
  hipMemcpy(train_spvdo_cu, train_spvdo_h, sizeof(float *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(train_spvin_cu, train_spvin_h, sizeof(int *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(train_n_cu, train_n, sizeof(int)*userCount, hipMemcpyHostToDevice);
  float *wi_cu;                                                                 
  hipMalloc((void**)&wi_cu, sizeof(float)*itemCount);                         
  hipMemcpy(wi_cu, Wi, sizeof(float)*itemCount, hipMemcpyHostToDevice);   
  
  cout<<"cuda: "<<endl;
  updateUserCuda2<<<BLOCK_NUM,THREAD_NUM>>>(prediction_items, rating_items, w_items, userCount, factors, reg, w_cu, u_cu, v_cu, wi_cu, sv_cu, train_spvdo_cu, train_n_cu, train_spvin_cu, v_col);
  
  //cout<<"cuda: "<<endl;
  hipMemcpy(u_h, u_cu, sizeof(float *)*userCount, hipMemcpyDeviceToHost); //?    
  //int dif = 0;                                                                  
  for (int u = 0; u < userCount; u++){                                                           
    hipMemcpy(U[u], u_h[u], sizeof(float)*factors, hipMemcpyDeviceToHost);
    for(int i=0; i<factors; i++)
      cout<<U[u][i]<<" ";
  }                         
  cout<<endl;
   
  for(int u=0; u<userCount; u++){
    hipFree(t1[u]);
    hipFree(t2[u]);
    hipFree(t3[u]);
    hipFree(t4[u]);
  }  
  free(t1); 
  free(t2);
  free(t3);
  free(t4);                        
  hipFree(prediction_items);                                                   
  hipFree(rating_items);                                                       
  hipFree(w_items);                                                            
  hipFree(v_col);                                                              
  hipFree(train_n_cu);                                                         
  free(train_n);                                                                
  hipFree(wi_cu);                                                              
  for (int u = 0; u < userCount; u++){                                          
    hipFree(w_h[u]);                                                           
    hipFree(train_spvin_h[u]);                                                 
    hipFree(train_spvdo_h[u]);                                                 
    hipFree(u_h[u]);                                                           
  }                                                                             
  for (int u = 0; u < itemCount; u++){                                          
    hipFree(v_h[u]);                                                           
  }                                                                             
  for (int u = 0; u < factors; u++){                                            
    hipFree(sv_h[u]);                                                          
  }                                                                             
  hipFree(w_cu);                                                               
  hipFree(u_cu);                                                               
  hipFree(v_cu);                                                               
  hipFree(sv_cu);                                                              
  hipFree(train_spvin_cu);                                                     
  hipFree(train_spvdo_cu);                                                     
  free(w_h);                                                                    
  free(u_h);                                                                    
  free(v_h);                                                                    
  free(sv_h);                                                                   
  free(train_spvin_h);                                                          
  free(train_spvdo_h);              

}

void MF_fastALS::updateUserSchedule3(float **trainMatrixdo, float **W, float **U, int **trainMatrixin, float **V, float **SV, float *Wi){
  int userCount = 10;
  int itemCount = 20;
  int factors = 8;

  int max_size = 5, size;
  float **prediction_items;
  float **rating_items;
  float **w_items, **v_col;

  hipMalloc((void**)&prediction_items,sizeof(float *)*userCount);
  hipMalloc((void**)&rating_items,sizeof(float *)*userCount);
  hipMalloc((void**)&w_items,sizeof(float *)*userCount);
  hipMalloc((void**)&v_col,sizeof(float *)*userCount);

  float **t1, **t2, **t3, **t3;
  t1 = (float *)malloc(sizeof(float)*userCount);
  t2 = (float *)malloc(sizeof(float)*userCount);
  t3 = (float *)malloc(sizeof(float)*userCount);
  t4 = (float *)malloc(sizeof(float)*userCount);

  for(int u=0; u<userCount; u++){
    hipMalloc((void**)&t1[u],sizeof(float)*max_size);
    hipMalloc((void**)&t2[u],sizeof(float)*max_size);
    hipMalloc((void**)&t3[u],sizeof(float)*max_size);
    hipMalloc((void**)&t4[u],sizeof(float)*max_size);
  }
  hipMemcpy(prediction_items, t1[u], sizeof(float)*max_size, hipMemcpyHostToDevice);
  hipMemcpy(rating_items, t2[u], sizeof(float)*max_size, hipMemcpyHostToDevice);
  hipMemcpy(w_items, t3[u], sizeof(float)*max_size, hipMemcpyHostToDevice);
  hipMemcpy(v_col, t4[u], sizeof(float)*max_size, hipMemcpyHostToDevice);

  float **w_cu, **u_cu, **v_cu, **sv_cu, **train_spvdo_cu;
  int **train_spvin_cu;
  hipMalloc((void**)&w_cu,sizeof(float *)*userCount);
  hipMalloc((void**)&u_cu,sizeof(float *)*userCount);
  hipMalloc((void**)&v_cu,sizeof(float *)*itemCount);
  hipMalloc((void**)&sv_cu,sizeof(float *)*factors);
  hipMalloc((void**)&train_spvin_cu,sizeof(int *)*userCount);
  hipMalloc((void**)&train_spvdo_cu,sizeof(float *)*userCount);
  int *train_n, *train_n_cu;
  train_n = (int *)malloc(sizeof(int)*userCount);
  hipMalloc((void**)&train_n_cu,sizeof(int)*userCount);

  float **w_h, **u_h, **v_h, **sv_h, **train_spvdo_h;
  int **train_spvin_h;
  w_h = (float **)malloc(sizeof(float *)*userCount);
  u_h = (float **)malloc(sizeof(float *)*userCount);
  v_h = (float **)malloc(sizeof(float *)*itemCount);
  sv_h = (float **)malloc(sizeof(float *)*factors);
  train_spvin_h = (int **)malloc(sizeof(int *)*userCount);
  train_spvdo_h = (float **)malloc(sizeof(float *)*userCount);
  for (int u = 0; u < userCount; u++){
    size = 5;
    train_n[u] = size;
    float *tmp_train, *tmp_w, *tmp_u;
    int *tmp;
    hipMalloc((void**)&tmp_train,sizeof(float)*size);
    hipMalloc((void**)&tmp_w,sizeof(float)*size);
    hipMalloc((void**)&tmp_u,sizeof(float)*factors);
    hipMalloc((void**)&tmp,sizeof(int)*size);
    hipMemcpy(tmp_train, trainMatrixdo[u], sizeof(float)*size, hipMemcpyHostToDevice);
    hipMemcpy(tmp_w, W[u], sizeof(float)*size, hipMemcpyHostToDevice);
    hipMemcpy(tmp_u, U[u], sizeof(float)*factors, hipMemcpyHostToDevice);
    hipMemcpy(tmp, trainMatrixin[u], sizeof(int)*size, hipMemcpyHostToDevice);

    w_h[u] = tmp_w;
    u_h[u] = tmp_u;
    train_spvin_h[u] = tmp;
    train_spvdo_h[u] = tmp_train;
  }

  for (int u = 0; u < itemCount; u++){
    float *tmp_v;
    hipMalloc((void**)&tmp_v,sizeof(float)*factors);
    hipMemcpy(tmp_v, V[u], sizeof(float)*factors, hipMemcpyHostToDevice);
    v_h[u] = tmp_v;
  }
  for (int u = 0; u < factors; u++){
    float *tmp_sv;
    hipMalloc((void**)&tmp_sv,sizeof(float)*factors);
    hipMemcpy(tmp_sv, SV[u], sizeof(float)*factors, hipMemcpyHostToDevice);
    sv_h[u] = tmp_sv;
  }

  hipMemcpy(u_cu, u_h, sizeof(float *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(v_cu, v_h, sizeof(float *)*itemCount, hipMemcpyHostToDevice);
  hipMemcpy(sv_cu, sv_h, sizeof(float *)*factors, hipMemcpyHostToDevice);
  hipMemcpy(w_cu, w_h, sizeof(float *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(train_spvdo_cu, train_spvdo_h, sizeof(float *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(train_spvin_cu, train_spvin_h, sizeof(int *)*userCount, hipMemcpyHostToDevice);
  hipMemcpy(train_n_cu, train_n, sizeof(int)*userCount, hipMemcpyHostToDevice);
  float *wi_cu;
  hipMalloc((void**)&wi_cu, sizeof(float)*itemCount);
  hipMemcpy(wi_cu, Wi, sizeof(float)*itemCount, hipMemcpyHostToDevice);

  updateUserCuda<<<BLOCK_NUM,THREAD_NUM>>>(prediction_items, rating_items, w_items, userCount, factors, reg, w_cu, u_cu, v_cu, wi_cu, sv_cu, train_spvdo_cu, train_n_cu, train_spvin_cu, v_col);

  hipMemcpy(u_h, u_cu, sizeof(float *)*userCount, hipMemcpyDeviceToHost); //?
  int dif = 0;
  for (int u = 0; u < userCount; u++){
    hipMemcpy(U[u], u_h[u], sizeof(float)*factors, hipMemcpyDeviceToHost);
  }

  for(int u=0; u<userCount; u++){
    hipFree(t1[u]);
    hipFree(t2[u]);
    hipFree(t3[u]);
    hipFree(t4[u]);
  }
  free(t1);
  free(t2);
  free(t3);
  free(t4);
  hipFree(prediction_items);
  hipFree(rating_items);
  hipFree(w_items);
  hipFree(v_col);
  hipFree(train_n_cu);
  free(train_n);
  hipFree(wi_cu);
  for (int u = 0; u < userCount; u++){
    hipFree(w_h[u]);
    hipFree(train_spvin_h[u]);
    hipFree(train_spvdo_h[u]);
    hipFree(u_h[u]);
  }
  for (int u = 0; u < itemCount; u++){
    hipFree(v_h[u]);
  }
  for (int u = 0; u < factors; u++){
    hipFree(sv_h[u]);
  }
  hipFree(w_cu);
  hipFree(u_cu);
  hipFree(v_cu);
  hipFree(sv_cu);
  hipFree(train_spvin_cu);
  hipFree(train_spvdo_cu);
  free(w_h);
  free(u_h);
  free(v_h);
  free(sv_h);
  free(train_spvin_h);
  free(train_spvdo_h);

}

*/
/*
__global__ void computeSUValue (float *u_clone, float *u, int f, int k, int factors,  float *result, int userCount){
  __shared__ float cache[THREAD_NUM];
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int cacheIndex = threadIdx.x;

  float temp = 0;
  int index;
  for(int i = bid*THREAD_NUM+tid; i < userCount; i+=BLOCK_NUM*THREAD_NUM){
    index = i * factors;
    temp = temp - u_clone[index+f] * u_clone[index+k] + u[index+f] * u[index+k];
  }

  cache[cacheIndex] = temp;
  __syncthreads();

  int i = blockDim.x/2;
  while (i != 0) {
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2;
  }
  if (cacheIndex == 0)
    result[blockIdx.x] = cache[0];
}

void testUpdateSu(){
  int userCount = 10;
  int factors = 5;

  float U[userCount][factors];
  float u_clone[userCount][factors];
  float res_cpp[factors][factors];
  float res_cuda[factors][factors];

  for(int i=0; i<userCount; i++){
    for(int j=0; j<factors; j++){
      U[i][j] = i;
      u_clone[i][j] = i+j;
    }
  }

  //cpp version
  float tmp;
    for (int f = 0; f < factors; f++) {                                                                                           
	    for (int k = 0; k <= f; k++) {
        float val = 0;
	      #pragma omp parallel for reduction(+:val) 
	      for (int u = 0; u < userCount; u++){
		      tmp = 0 - u_clone[u][f] * u_clone[u][k] + U[u][f] * U[u][k];
          val += tmp;
        }
		    res_cpp[f][k] = val;                                                      
		    res_cpp[k][f] = val;                                                                                                                                 
      }     
    }

  //cuda version
  int byteSize = sizeof(float)*userCount*factors;
  float *U_clone_cu, *U_cu, *U_h, *U_clone_h, *result, *result_cu;
  U_h = (float *)malloc(byteSize);
  U_clone_h = (float *)malloc(byteSize);
  result = (float *)malloc(sizeof(float)*(BLOCK_NUM));
  hipMalloc((void**)&U_clone_cu, byteSize);
  hipMalloc((void**)&U_cu, byteSize);
  hipMalloc((void**)&result_cu, sizeof(float)*(BLOCK_NUM));
  int ii = 0;

  #pragma omp parallel for
  for (int u = 0; u < userCount; u++){
    for(int j = 0; j < factors; j++){
      U_h[ii] = U[u][j];
      U_clone_h[ii] = u_clone[u][j];
      ii++;
    }
  }

  hipMemcpy(U_clone_cu, U_clone_h, byteSize, hipMemcpyHostToDevice);
  hipMemcpy(U_cu, U_h, byteSize, hipMemcpyHostToDevice);
    
    //int indexf, indexk;
    for (int f = 0; f < factors; f++) { 
      for (int k = 0; k <= f; k++) {                                          
        float val = 0;
        computeSUValue<<<BLOCK_NUM,THREAD_NUM,0>>>(U_clone_cu, U_cu, f, k, factors, result_cu, userCount);
        hipMemcpy(result, result_cu, sizeof(float)*(BLOCK_NUM), hipMemcpyDeviceToHost);
        #pragma omp parallel for reduction(+:val)                          
        for (int u = 0; u < BLOCK_NUM; u++){                                  
          val += result[u];                                                       
        }                                                                     
        res_cuda[f][k] = val;
        res_cuda[k][f] = val;
       }                                                                  
    }

    hipFree(U_clone_cu);
    hipFree(U_cu);
    hipFree(result_cu);
    hipHostFree(U_clone_h);
    hipHostFree(U_h);
    hipHostFree(result);

    for(int i=0; i<factors; i++){
      for(int j=0; j<factors; j++){
        std::cout<<"i: "<<i<<" j: "<<j<<" cpp: "<<res_cpp[i][j]<<" cuda: "<<res_cuda[i][j]<<endl;
      }
    }
}
*/
/*
__global__ void computeSVValue (float *u_clone, float *u, int f, int k, int factors, float *result, int userCount, float *wi_cu){
  __shared__ float cache[THREAD_NUM];
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int cacheIndex = threadIdx.x;

  float temp = 0;
  float val = 0;
  int index;
  for(int i = bid*THREAD_NUM+tid; i < userCount; i+=BLOCK_NUM*THREAD_NUM){
    index = i * factors;
    temp = 0 - u_clone[index+f] * u_clone[index+k] + u[index+f] * u[index+k];
    //printf("%f %f %f %f\n", u_clone[index+f], u_clone[index+k], u[index+f], u[index+k]);
    //printf("%f ", temp);
    val += temp*wi_cu[i];
  }

  cache[cacheIndex] = val;
  __syncthreads();

  int i = blockDim.x/2;
  while (i != 0) {
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2;
  }
  if (cacheIndex == 0)
    result[blockIdx.x] = cache[0];
  
}

void testUpdateSv(){
  int itemCount = 10;
  int factors = 5;
  float V[itemCount][factors];
  float v_clone[itemCount][factors];
  float wii[itemCount];
  float res_cpp[factors][factors];
  float res_cuda[factors][factors];

  for(int i=0; i<itemCount; i++){
    for(int j=0; j<factors; j++){
      V[i][j] = 2;
      v_clone[i][j] = 1;
    }
    wii[i] = 0.5;
  }

  //cpp version
  float tmp;
    for (int f = 0; f < factors; f++) {
	    for (int k = 0; k <= f; k++) {
		    float val = 0;
 		    #pragma omp parallel for reduction(+:val)
  		  for (int u = 0; u < itemCount; u++){
 			    tmp = 0 - v_clone[u][f] * v_clone[u][k] + V[u][f] * V[u][k] ;
           //cout<<tmp<<" ";
           tmp = tmp *  wii[u];
          val += tmp;
          //cout<<v_clone[u][f]<<" "<<v_clone[u][k]<<" "<<V[u][f]<<" "<<V[u][k]<<"\n";
          //cout<<tmp<<" ";
		    }
        res_cpp[f][k] = val;
        res_cpp[k][f] = val;
      }
    }
   //cout<<"cpp is okay"<<endl; 

  int byteSize = sizeof(float)*itemCount*factors;
    float *V_clone_cu, *V_cu, *V_h, *V_clone_h, *resultv, *resultv_cu, *wi_cu;
    V_h = (float *)malloc(byteSize);
    V_clone_h = (float *)malloc(byteSize);
    resultv = (float *)malloc(sizeof(float)*(BLOCK_NUM));
    hipMalloc((void**)&V_clone_cu, byteSize);
    hipMalloc((void**)&V_cu, byteSize);
    hipMalloc((void**)&wi_cu, sizeof(float)*itemCount);
    hipMalloc((void**)&resultv_cu, sizeof(float)*(BLOCK_NUM));
    int ii = 0;

    #pragma omp parallel for
    for (int u = 0; u < itemCount; u++){
      for(int j = 0; j < factors; j++){
        V_h[ii] = V[u][j];
        V_clone_h[ii] = v_clone[u][j];
        ii++;
      }
    }
   //cout<<"1005 is okay"<<endl;
    hipMemcpy(V_clone_cu, V_clone_h, byteSize, hipMemcpyHostToDevice);
    hipMemcpy(V_cu, V_h, byteSize, hipMemcpyHostToDevice);
    hipMemcpy(wi_cu, wii, sizeof(float)*itemCount, hipMemcpyHostToDevice);

    for (int f = 0; f < factors; f++) {
      for (int k = 0; k <= f; k++) {
        float val = 0;
        computeSVValue<<<BLOCK_NUM,THREAD_NUM,0>>>(V_clone_cu, V_cu, f, k, factors, resultv_cu, itemCount, wi_cu);
        hipMemcpy(resultv, resultv_cu, sizeof(float)*(BLOCK_NUM), hipMemcpyDeviceToHost);
        #pragma omp parallel for reduction(+:val)
        for (int u = 0; u < BLOCK_NUM; u++){
          val += resultv[u];
        } 
        //cout<<val<<endl;
        res_cuda[f][k] = val;
        res_cuda[k][f] = val;
      }
    }
    //cout<<"1023 is okay"<<endl;
    hipFree(V_clone_cu);
    hipFree(V_cu);
    hipFree(resultv_cu);
    hipFree(wi_cu);
    free(V_h);
    free(V_clone_h);
    free(resultv);
    
    for(int i=0; i<factors; i++){
      for(int j=0; j<factors; j++){
        if(res_cpp[i][j]!=res_cuda[i][j])
          std::cout<<"i: "<<i<<" j: "<<j<<" cpp: "<<res_cpp[i][j]<<" cuda: "<<res_cuda[i][j]<<endl;
      }
    }
    
}
*/

void updateUserCpp(float *prediction_items, float *rating_items, float *w_items, int userCount, int factors, float reg, float **w_cu1, float **u_cu, float **v_cu, float *wi_cu, float **sv_cu, float **train_spvdo_cu, int *train_n_cu, int **train_spvin_cu, float *v_col){
  int size_item;
  float ifv, ufget, tmp_uget;
  int i;
  float res;
  float numer, denom;
  int *itemList;
  for(int u = 0; u < userCount; u++){
    itemList = train_spvin_cu[u];
    size_item = train_n_cu[u];
    if (size_item == 0)        continue ;
    for (int j = 0; j < size_item; j++) {
      i = itemList[j];
      res = 0;
      for(int k=0; k<factors; k++){
        res += u_cu[u][k] * v_cu[i][k];
      }
      prediction_items[j] = res;
      rating_items[j] = train_spvdo_cu[u][j];
      w_items[j] = w_cu1[u][j];
    }
    for (int f = 0; f < factors; f++) {
      numer = 0, denom = 0;
      for(int j = 0; j<size_item; j++){
        i = itemList[j];
        v_col[j] = v_cu[i][f];
      }
      for(int k = 0; k<factors; k++){
        if(k!=f){
          numer -= u_cu[u][k] * sv_cu[f][k];
          //cout<<u_cu[u][k]<<" ";
        }
      }
      //cout<<endl;
      //cout<<numer<<" ";
      ufget = u_cu[u][f];
      for (int j = 0; j<size_item; j++) {
        i = itemList[j];
        ifv = *(v_col+j);
        prediction_items[j] -= ufget * ifv;
        numer += (w_items[j] * rating_items[j] - (w_items[j] - wi_cu[i]) * prediction_items[j]) * ifv;
        denom += (w_items[j] - wi_cu[i]) * ifv * ifv;
        /*if(u==userCount-1){
          printf("%f ", w_items[j]);
        }*/
      }
      //cout<<denom<<" ";
      denom +=sv_cu[f][f] + reg;
      //cout<<numer<<" ";
      u_cu[u][f] = numer / denom;
      //cout<<"numer: "<<numer<<"denom: "<<denom<<" ";
      tmp_uget = numer / denom;
      for (int j = 0; j<size_item; j++){
        prediction_items[j] += tmp_uget * v_col[j];
      }
    }
  }
}

void updateUserSchedule1(){
int userCount = 10;
int itemCount = 20;
int factors = 8;
float reg = 0;

int max_size = 5, size;
float *prediction_items;
float *rating_items;
float *w_items, *v_col;
prediction_items = (float *)malloc(sizeof(float)*max_size);
rating_items = (float *)malloc(sizeof(float)*max_size);
w_items = (float *)malloc(sizeof(float)*max_size);
v_col = (float *)malloc(sizeof(float)*max_size);                                                                    

float **w_cu, **u_cu, **v_cu, **sv_cu, **train_spvdo_cu, **u2;
int **train_spvin_cu;
w_cu = (float **)malloc(sizeof(float *)*userCount);
u_cu = (float **)malloc(sizeof(float *)*userCount);
u2 = (float **)malloc(sizeof(float *)*userCount);
v_cu = (float **)malloc(sizeof(float *)*itemCount);
sv_cu = (float **)malloc(sizeof(float *)*factors);
train_spvin_cu = (int **)malloc(sizeof(int *)*userCount);
train_spvdo_cu = (float **)malloc(sizeof(float *)*userCount);

int *train_n_cu;
train_n_cu = (int *)malloc(sizeof(int)*userCount);

for (int u = 0; u < userCount; u++){
  size = 5;
  w_cu[u] = (float *)malloc(sizeof(float)*size);
  train_spvin_cu[u] = (int *)malloc(sizeof(int)*size);
  train_spvdo_cu[u] = (float *)malloc(sizeof(float)*size);
  for(int i = 0; i < size; i++){
    w_cu[u][i] = 0.5;
    train_spvin_cu[u][i] = i;
    train_spvdo_cu[u][i] = 1;
  }
  train_n_cu[u] = size;
  u_cu[u] = (float *)malloc(sizeof(float)*factors);
  u2[u] = (float *)malloc(sizeof(float)*factors);
  for(int i = 0; i < factors; i++){
    u_cu[u][i] = i;
    u2[u][i] = u_cu[u][i];
  }
  }


  for (int u = 0; u < itemCount; u++){
    v_cu[u] = (float *)malloc(sizeof(float)*factors);
    for(int i = 0; i < factors; i++)
      v_cu[u][i] = 1;
  }
  for (int u = 0; u < factors; u++){
    sv_cu[u] = (float *)malloc(sizeof(float)*factors);
    for(int i = 0; i < factors; i++)
      sv_cu[u][i] = 2;
  }

  float *wi_cu;
  wi_cu = (float *)malloc(sizeof(float)*itemCount);
  for(int i=0; i<itemCount; i++){
    wi_cu[i] = ((float)i)/5;
  }

  updateUserCpp(prediction_items, rating_items, w_items, userCount, factors, reg, w_cu, u_cu, v_cu, wi_cu, sv_cu, train_spvdo_cu, train_n_cu, train_spvin_cu, v_col);

  cout<<"cpp: "<<endl;
  //float sum = 0;
  for(int u=0; u<userCount; u++){
    for(int i=0; i<factors; i++)
    {   cout<<u_cu[u][i]<<" ";
        //sum += u_cu[u][i];
    }
  }
  cout<<endl;
  free(wi_cu);
  free(prediction_items);
  free(rating_items);
  free(w_items);
  free(v_col);
  free(train_n_cu);
  for (int u = 0; u < userCount; u++){
    free(w_cu[u]);
    free(train_spvin_cu[u]);
    free(train_spvdo_cu[u]);
    free(u_cu[u]);
    free(u2[u]);
  }
  for (int u = 0; u < itemCount; u++){
    free(v_cu[u]);
  }
  for (int u = 0; u < factors; u++){
    free(sv_cu[u]);
  }
  
  free(w_cu);
  free(u_cu);
  free(u2);
  free(v_cu);
  free(sv_cu);
  free(train_spvin_cu);                                                                                               
  free(train_spvdo_cu);
}

__global__ void updateUserCuda(float *numer, float *denom, float *v_col, int userCount, int factors, float reg, float *w_cu, float *v_cu, float *sv_cu, float *wi_cu, float *train_spvdo_cu, int *train_n_cu, int *train_spvin_cu){
  int tidx = threadIdx.x;
  //int tidy = threadIdx.y;
  int bidx = blockIdx.x;
  int bidy = blockIdx.y;
  int size_item;
  float ifv;
  int i, index_u;
  int size2;
  int index;
  float ww;
  __shared__ float numer_sh[2];
  __shared__ float denom_sh[2];

  for(int u = bidx; u < userCount; u+=gridDim.x){
    size_item = train_n_cu[u+1] - train_n_cu[u];
    size2 = train_n_cu[u];
    if (size_item == 0)        continue ;
    for (int f = bidy; f < factors; f+=gridDim.y) {
      index_u = u*factors+f;
      numer_sh[tidx] = 0;
      denom_sh[tidx] = 0;
      for (int j = tidx; j < size_item; j+=blockDim.x) {
        index = size2+j;
        i = train_spvin_cu[index];
        ifv = v_cu[i*factors+f];
        ww = w_cu[index];
        numer_sh[tidx] += ww * train_spvdo_cu[index] * ifv;
        denom_sh[tidx] += (ww - wi_cu[i]) * ifv * ifv;
      }
      __syncthreads();
      i = blockDim.x/2;
      while (i != 0) {
        if (tidx < i){
          numer_sh[tidx] += numer_sh[tidx + i];
          denom_sh[tidx] += denom_sh[tidx + i];
        }  
        __syncthreads();
        i /= 2;
      }
      if(tidx == 0) {
        denom[index_u] = denom_sh[0] + sv_cu[f*factors+f] + reg;
        numer[index_u] = numer_sh[0];
      }
      __syncthreads();
      //numer[index_u] += uvsv_cu[u*factors+f] * uvsv_cu[vborder+f*factors+f];
      //denom[index_u] += sv_cu[f*factors+f] + reg;
      //printf("%f ", numer[index_u]);
    }
  }
}

void updateUserSchedule(){
  int size, size2;

  //initial
  int userCount = 10;
  int itemCount = 20;
  int factors = 8;
  float reg = 0;
  size = 5;
  float trainMatrix_n[userCount];
  float W[userCount][size];
  float trainMatrix_spvin[userCount][size];
  float trainMatrix_spvdo[userCount][size];
  float U[userCount][factors];
  float V[itemCount][factors];
  float SV[factors][factors];
  for (int u = 0; u < userCount; u++){
    trainMatrix_n[u] = size;
    for(int i = 0; i < size; i++){
      W[u][i] = 0.5;
      trainMatrix_spvin[u][i] = i;
      trainMatrix_spvdo[u][i] = 1;
    }
    for(int i = 0; i < factors; i++){
      U[u][i] = (float)i;
    }
  }
  for (int u = 0; u < itemCount; u++){
    for(int i = 0; i < factors; i++)
      V[u][i] = 1;
  }
  for (int u = 0; u < factors; u++){
    for(int i = 0; i < factors; i++)
      SV[u][i] = 2;
  }

  float Wi[itemCount];
  for(int i=0; i<itemCount; i++){
    Wi[i] = ((float)i)/5;
  }

  
  float prediction_items[size], *v_col;
  //int total_size = trainMatrix.itemCount()+10;
  int total_size = 50+10;
  //hipMalloc((void**)&prediction_items,sizeof(float)*total_size);
  hipMalloc((void**)&v_col,sizeof(float)*total_size);

  float *w_cu, *u_cu, *v_cu, *sv_cu, *train_spvdo_cu;
  int *train_spvin_cu;
  hipMalloc((void**)&w_cu, sizeof(float)*total_size);
  //hipMalloc((void**)&u_cu, sizeof(float)*userCount*factors);
  hipMalloc((void**)&v_cu, sizeof(float)*itemCount*factors);
  hipMalloc((void**)&sv_cu, sizeof(float)*factors*factors);
  hipMalloc((void**)&train_spvin_cu,sizeof(int)*total_size);
  hipMalloc((void**)&train_spvdo_cu,sizeof(float)*total_size);
  int *train_n, *train_n_cu;
  hipMalloc((void**)&train_n_cu,sizeof(int)*(userCount+1));
  float *w_h, *u_h, *v_h, *sv_h, *train_spvdo_h;
  int *train_spvin_h;
  
  hipHostAlloc((void**)&w_h, sizeof(float)*total_size, hipHostMallocDefault);
  //hipHostAlloc((void**)&u_h, sizeof(float)*userCount*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&v_h, sizeof(float)*itemCount*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&sv_h, sizeof(float)*factors*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&train_spvin_h, sizeof(int)*total_size, hipHostMallocDefault);
  hipHostAlloc((void**)&train_spvdo_h, sizeof(float)*total_size, hipHostMallocDefault);
  hipHostAlloc((void**)&train_n, sizeof(int)*(userCount+1), hipHostMallocDefault);
  
  train_n[0] = 0;
  for (int u = 0; u < userCount; u++){
    size = trainMatrix_n[u];
    size2 = train_n[u];
    train_n[u+1] = size2 + size;

    for(int i=0; i<size; i++){
      w_h[size2+i] = W[u][i];
      train_spvin_h[size2+i] = trainMatrix_spvin[u][i];
      train_spvdo_h[size2+i] = trainMatrix_spvdo[u][i];
    }
  }
  /*
  for (int u = 0; u < userCount; u++){
    for(int i=0; i<factors; i++)
      u_h[u*factors+i] = U[u][i];
  }
  */
  for (int u = 0; u < itemCount; u++){
    for(int i=0; i<factors; i++)
      v_h[u*factors+i] = V[u][i];
  }
  for (int u = 0; u < factors; u++){
    for(int i=0; i<factors; i++)
      sv_h[u*factors+i] = SV[u][i];
  }
  //hipMemcpy(u_cu, u_h, sizeof(float)*userCount*factors, hipMemcpyHostToDevice);
  hipMemcpy(v_cu, v_h, sizeof(float)*itemCount*factors, hipMemcpyHostToDevice);
  hipMemcpy(sv_cu, sv_h, sizeof(float)*factors*factors, hipMemcpyHostToDevice);
  hipMemcpy(w_cu, w_h, sizeof(float)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_spvdo_cu, train_spvdo_h, sizeof(float)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_spvin_cu, train_spvin_h, sizeof(int)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_n_cu, train_n, sizeof(int)*(userCount+1), hipMemcpyHostToDevice);
  float *wi_cu;
  hipMalloc((void**)&wi_cu, sizeof(float)*itemCount);
  hipMemcpy(wi_cu, Wi, sizeof(float)*itemCount, hipMemcpyHostToDevice);

  float *u_numer, *u_denom;
  hipMalloc((void**)&u_numer,sizeof(float)*userCount*factors);
  hipMalloc((void**)&u_denom,sizeof(float)*userCount*factors);

  int dimx, dimy;
  dimx = 2;
  dimy = 2;
  //dim3 block(dimx, dimy);
  //dim3 grid(2);
  dim3 block(2);
  dim3 grid(dimx, dimy);
  updateUserCuda<<<grid, block>>>(u_numer, u_denom, v_col, userCount, factors, reg, w_cu, v_cu, sv_cu, wi_cu, train_spvdo_cu, train_n_cu, train_spvin_cu);
  float u_numer_h[userCount*factors];
  float u_denom_h[userCount*factors];
  hipMemcpy(u_numer_h, u_numer, sizeof(float)*userCount*factors, hipMemcpyDeviceToHost);
  hipMemcpy(u_denom_h, u_denom, sizeof(float)*userCount*factors, hipMemcpyDeviceToHost);
  //updagte prediction_items and U
  float res, tmp_uget, ufget, ifv;
  int i, size_item;
  for(int u = 0; u < userCount; u++){
    size_item = trainMatrix_n[u];
    for (int j = 0; j < size_item; j++) {
      i = trainMatrix_spvin[u][j];
      res = 0;
      for(int k=0; k<factors; k++){
        res += U[u][k] * V[i][k];
      }
      prediction_items[j] = res;
    }
    for (int f = 0; f < factors; f++) {
      ufget = U[u][f];
      for(int k = 0; k<factors; k++){
        if(k!=f){
          u_numer_h[u*factors+f] -= U[u][k] * SV[f][k];
        }
      }
      for (int j = 0; j<size_item; j++) {
        i = trainMatrix_spvin[u][j];
        ifv = V[i][f];
        prediction_items[j] -= ufget * ifv;
        u_numer_h[u*factors+f] += (Wi[i] - W[u][j]) * prediction_items[j] * ifv;
      }
      U[u][f] = u_numer_h[u*factors+f] / u_denom_h[u*factors+f];
      //cout<<"numer: "<<u_numer_h[u*factors+f]<<"denom: "<<u_denom_h[u*factors+f]<<" ";
      //cout<<u_numer_h[u*factors+f]<<" ";
      tmp_uget = U[u][f];
      for (int j = 0; j<size_item; j++){
        i = trainMatrix_spvin[u][j];
        prediction_items[j] += tmp_uget * V[i][f];
      }
    }
  }

  cout<<"cuda: \n";
  for (int u = 0; u < userCount; u++){
    for(int i=0; i<factors; i++){
      //U[u][i] = uvsv_h[u*factors+i];
      cout<<U[u][i]<<" ";
    }
  }
  cout<<endl;
  //hipFree(prediction_items);
  hipFree(v_col);
  hipFree(train_n_cu);
  hipFree(w_cu);
  //hipFree(u_cu);
  hipFree(v_cu);
  hipFree(sv_cu);
  hipFree(train_spvin_cu);
  hipFree(train_spvdo_cu);
  hipFree(wi_cu);
  hipFree(u_numer);
  hipFree(u_denom);
  hipHostFree(train_n);
  hipHostFree(w_h);
  //hipHostFree(u_h);
  hipHostFree(v_h);
  hipHostFree(sv_h);
  hipHostFree(train_spvin_h);
  hipHostFree(train_spvdo_h);
}

__global__ void updateUserCuda2(float *numer, float *denom, float *prediction_items, int userCount, int factors, float reg, float *w_cu, float *u_cu, float *v_cu, float *sv_cu, float *wi_cu, float *train_spvdo_cu, int *train_n_cu, int *train_spvin_cu){
  int tidx = threadIdx.x;
  //int tidy = threadIdx.y;
  int bidx = blockIdx.x;
  int bidy = blockIdx.y;
  int size_item;
  float ifv, ufget;
  int i, index_u;
  int size2;
  int index;
  float ww, res, tmp_uget;
  __shared__ float numer_sh[2];
  __shared__ float denom_sh[2];

  for(int u = bidx; u < userCount; u+=gridDim.x){
    size_item = train_n_cu[u+1] - train_n_cu[u];
    size2 = train_n_cu[u];
    if (size_item == 0)        continue ;
    for (int f = bidy; f < factors; f+=gridDim.y) {
      index_u = u*factors+f;
      numer_sh[tidx] = 0;
      denom_sh[tidx] = 0;
      for (int j = tidx; j < size_item; j+=blockDim.x) {
        index = size2+j;
        i = train_spvin_cu[index];
        ifv = v_cu[i*factors+f];
        ww = w_cu[index];
        numer_sh[tidx] += ww * train_spvdo_cu[index] * ifv;
        denom_sh[tidx] += (ww - wi_cu[i]) * ifv * ifv;
      }
      __syncthreads();
      i = blockDim.x/2;
      while (i != 0) {
        if (tidx < i){
          numer_sh[tidx] += numer_sh[tidx + i];
          denom_sh[tidx] += denom_sh[tidx + i];
        }  
        __syncthreads();
        i /= 2;
      }
      if(tidx == 0) {
        denom[index_u] = denom_sh[0] + sv_cu[f*factors+f] + reg;
        numer[index_u] = numer_sh[0];
      }
      __syncthreads();
    }
  }

  int threadId = tidx + blockDim.x * (bidy * gridDim.x + bidx);
  int dist = gridDim.x * gridDim.y * blockDim.x;
  for(int u = threadId; u < userCount; u+=dist){
    size_item = train_n_cu[u+1] - train_n_cu[u];
    size2 = train_n_cu[u];
    if (size_item == 0)        continue ;
    for (int j = 0; j < size_item; j++) {
      index = size2+j;
      i = train_spvin_cu[index];
      res = 0;
      for(int k=0; k<factors; k++){
        res += u_cu[u*factors+k] * v_cu[i*factors+k];
      }
      prediction_items[index] = res;
    }
    for (int f = 0; f < factors; f++) {
      index_u = u*factors+f;
      ufget = u_cu[index_u];
      for (int j = 0; j < size_item; j++) {
          index = size2+j;
          i = train_spvin_cu[index];
          ifv = v_cu[i*factors+f];
          prediction_items[index] -= ufget * ifv;
          numer[index_u] += (wi_cu[i] - w_cu[index]) * prediction_items[index] * ifv;
        }
        for(int k = 0; k<factors; k++){
          numer[index_u] -= u_cu[u*factors+k] * sv_cu[f*factors+k];
        }
        numer[index_u] += u_cu[index_u] * sv_cu[f*factors+f];
        //printf("%f ", numer[u*factors+f]);
        u_cu[index_u] = numer[index_u] / denom[index_u];
        tmp_uget = u_cu[index_u];
        for (int j = 0; j < size_item; j++){
          i = train_spvin_cu[size2+j];
          prediction_items[size2+j] += tmp_uget * v_cu[i*factors+f];
        }
      }
    }
}

void updateUserSchedule2(){
  int size, size2;

  //initial
  int userCount = 10;
  int itemCount = 20;
  int factors = 8;
  float reg = 0;
  size = 5;
  float trainMatrix_n[userCount];
  float W[userCount][size];
  float trainMatrix_spvin[userCount][size];
  float trainMatrix_spvdo[userCount][size];
  float U[userCount][factors];
  float V[itemCount][factors];
  float SV[factors][factors];
  for (int u = 0; u < userCount; u++){
    trainMatrix_n[u] = size;
    for(int i = 0; i < size; i++){
      W[u][i] = 0.5;
      trainMatrix_spvin[u][i] = i;
      trainMatrix_spvdo[u][i] = 1;
    }
    for(int i = 0; i < factors; i++){
      U[u][i] = (float)i;
    }
  }
  for (int u = 0; u < itemCount; u++){
    for(int i = 0; i < factors; i++)
      V[u][i] = 1;
  }
  for (int u = 0; u < factors; u++){
    for(int i = 0; i < factors; i++)
      SV[u][i] = 2;
  }

  float Wi[itemCount];
  for(int i=0; i<itemCount; i++){
    Wi[i] = ((float)i)/5;
  }

  
  float *v_col;
  //int total_size = trainMatrix.itemCount()+10;
  int total_size = 50+10;
  hipMalloc((void**)&v_col,sizeof(float)*total_size);

  float *w_cu, *u_cu, *v_cu, *sv_cu, *train_spvdo_cu;
  int *train_spvin_cu;
  hipMalloc((void**)&w_cu, sizeof(float)*total_size);
  hipMalloc((void**)&u_cu, sizeof(float)*userCount*factors);
  hipMalloc((void**)&v_cu, sizeof(float)*itemCount*factors);
  hipMalloc((void**)&sv_cu, sizeof(float)*factors*factors);
  hipMalloc((void**)&train_spvin_cu,sizeof(int)*total_size);
  hipMalloc((void**)&train_spvdo_cu,sizeof(float)*total_size);
  int *train_n, *train_n_cu;
  hipMalloc((void**)&train_n_cu,sizeof(int)*(userCount+1));
  float *w_h, *u_h, *v_h, *sv_h, *train_spvdo_h;
  int *train_spvin_h;
  hipHostAlloc((void**)&w_h, sizeof(float)*total_size, hipHostMallocDefault);
  hipHostAlloc((void**)&u_h, sizeof(float)*userCount*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&v_h, sizeof(float)*itemCount*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&sv_h, sizeof(float)*factors*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&train_spvin_h, sizeof(int)*total_size, hipHostMallocDefault);
  hipHostAlloc((void**)&train_spvdo_h, sizeof(float)*total_size, hipHostMallocDefault);
  hipHostAlloc((void**)&train_n, sizeof(int)*(userCount+1), hipHostMallocDefault);
  
  train_n[0] = 0;
  for (int u = 0; u < userCount; u++){
    size = trainMatrix_n[u];
    size2 = train_n[u];
    train_n[u+1] = size2 + size;

    for(int i=0; i<size; i++){
      w_h[size2+i] = W[u][i];
      train_spvin_h[size2+i] = trainMatrix_spvin[u][i];
      train_spvdo_h[size2+i] = trainMatrix_spvdo[u][i];
    }
  }
  
  for (int u = 0; u < userCount; u++){
    for(int i=0; i<factors; i++)
      u_h[u*factors+i] = U[u][i];
  }
  
  for (int u = 0; u < itemCount; u++){
    for(int i=0; i<factors; i++)
      v_h[u*factors+i] = V[u][i];
  }
  for (int u = 0; u < factors; u++){
    for(int i=0; i<factors; i++)
      sv_h[u*factors+i] = SV[u][i];
  }
  hipMemcpy(u_cu, u_h, sizeof(float)*userCount*factors, hipMemcpyHostToDevice);
  hipMemcpy(v_cu, v_h, sizeof(float)*itemCount*factors, hipMemcpyHostToDevice);
  hipMemcpy(sv_cu, sv_h, sizeof(float)*factors*factors, hipMemcpyHostToDevice);
  hipMemcpy(w_cu, w_h, sizeof(float)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_spvdo_cu, train_spvdo_h, sizeof(float)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_spvin_cu, train_spvin_h, sizeof(int)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_n_cu, train_n, sizeof(int)*(userCount+1), hipMemcpyHostToDevice);
  float *wi_cu;
  hipMalloc((void**)&wi_cu, sizeof(float)*itemCount);
  hipMemcpy(wi_cu, Wi, sizeof(float)*itemCount, hipMemcpyHostToDevice);

  float *u_numer, *u_denom;
  hipMalloc((void**)&u_numer,sizeof(float)*userCount*factors);
  hipMalloc((void**)&u_denom,sizeof(float)*userCount*factors);

  int dimx, dimy;
  dimx = 2;
  dimy = 2;
  //dim3 block(dimx, dimy);
  //dim3 grid(2);
  dim3 block(2);
  dim3 grid(dimx, dimy);
  updateUserCuda2<<<grid, block>>>(u_numer, u_denom, v_col, userCount, factors, reg, w_cu, u_cu, v_cu, sv_cu, wi_cu, train_spvdo_cu, train_n_cu, train_spvin_cu);
  //float u_numer_h[userCount*factors];
  //float u_denom_h[userCount*factors];
  //hipMemcpy(u_numer_h, u_numer, sizeof(float)*userCount*factors, hipMemcpyDeviceToHost);
  //hipMemcpy(u_denom_h, u_denom, sizeof(float)*userCount*factors, hipMemcpyDeviceToHost);
  hipMemcpy(u_h, u_cu, sizeof(float)*userCount*factors, hipMemcpyDeviceToHost);
  
  cout<<"cuda: \n";
  for (int u = 0; u < userCount; u++){
    for(int i=0; i<factors; i++){
      U[u][i] = u_h[u*factors+i];
      cout<<U[u][i]<<" ";
    }
  }
  cout<<endl;
  hipFree(v_col);
  hipFree(train_n_cu);
  hipFree(w_cu);
  hipFree(u_cu);
  hipFree(v_cu);
  hipFree(sv_cu);
  hipFree(train_spvin_cu);
  hipFree(train_spvdo_cu);
  hipFree(wi_cu);
  hipFree(u_numer);
  hipFree(u_denom);
  hipHostFree(train_n);
  hipHostFree(w_h);
  hipHostFree(u_h);
  hipHostFree(v_h);
  hipHostFree(sv_h);
  hipHostFree(train_spvin_h);
  hipHostFree(train_spvdo_h);
}

//kernel computation
__global__ void updateUserCuda(int *tile_start, int wid_num, int hei_num, int tile_width, float *prediction_items, float *v_col, int userCount, int factors, float reg, float *w_cu, float *u_cu, float *v_cu, float *sv_cu, float *wi_cu, int *row_num, int *row_value_index, float *row_value){
	int tid = threadIdx.x;
	int start_u, end_u, start_i, end_i;
	start_u = (tile_start[tid]/wid_num)*tile_width;
	end_u = ((tile_start[tid+1]-1)/wid_num)*tile_width;
	//start_i = (tile_start[tid]/hei_num)*tile_width;
	//end_i = ((tile_start[tid+1]-1)/hei_num)*tile_width;
	if(tid<32){
		dim3 DimBlock1(32, 32);
		calculatePred<<<1，DimBlock1>>>(tile_start, tid, tile_width, start_u, end_u, u_cu, v_cu, row_num, row_value_index, row_value, prediction_items);
	}
	else{
		calculateNumerDenom<<<1, 64>>>(tile_start, tid, tile_width, start_u, end_u, u_cu, v_cu, w_cu, wi_cu, row_num, row_value_index, row_value, numer, denom);
	}
	__syncthreads();
	//dim3 DimBlock2(32, 32);
	calcualteU<<<1, 64>>>(tile_start, thread_id, tile_width, start_u, end_u, u_cu, v_cu, w_cu, wi_cu, row_num, row_value_index, row_value, numer, denom, prediction_items);
}


//matrix multiplication kernel
__device__ void calculatePred(int *tile_start, int thread_id, int tile_width, int start_u, int end_u, float *u_cu, float *v_cu, int *row_num, int *row_value_index, float *row_value, float *prediction_items){
	int tid_x = threadIdx.x;
	int tid_y = threadIdx.y;
	int index_i, length_start, length_end;
	int start_index;
	start_index = thread_id*tile_width;
	__shared__ float res[1024];
	for(int u=tid_x+start_u; u<tid_x+end_u; u++){
		length_start = row_num[start_index];
		length_end = row_num[start_index+1];
		for(int i=tid_y+length_start; i<tid_y+length_end; i++){
			res = 0;
			index_i = row_value_index[i];
			for(int k=0; k<factors; k++){
				//res += u_cu[u][k] * v_cu[index_i][k];
			//}
				res[tid_x*32+tid_y] = u_cu[u][k] * v_cu[index_i][k];
			}
			prediction_items[u][i-length_start] = res[tid_x*32+tid_y];
		}

		start_index++;
	}

}

//calcualte denom and numer
__device__ void calculateNumerDenom(int *tile_start, int thread_id, int tile_width, int start_u, int end_u, float *u_cu, float *v_cu, float *w_cu, float *wi_cu, int *row_num, int *row_value_index, float *row_value, float *numer, float *denom){
	int tid = threadIdx.x;
	int index_i, length_start, length_end;
	int start_index;
	start_index = thread_id*tile_width;
	__shared__ float numer_sh[64], denom_sh[64];
	__shared__ float tmp_values[3*64]
	for(int u=tid+start_u; u<tidend_u; u++){
		length_start = row_num[start_index];
		length_end = row_num[start_index+1];
		for(int k=0; k<factors; k++){
			numer[tid] = 0;
			denom[tid] = 0;
			for(int i=length_start; i<length_end; i++){
				index_i = row_value_index[i];
				tmp_values[tid*3] = w_cu[index_i];
				tmp_values[tid*3+1] = wi_cu[index_i];
				tmp_values[tid*3+2] = v_cu[index_i][k];
				numer_sh[tid] += tmp_values[tid*3] * row_value[i] * tmp_values[tid*3+2];
				denom_sh[tid] += (tmp_values[tid*3]- tmp_values[tid*3+1]) * tmp_values[tid*3+2] * tmp_values[tid*3+2];
			}
			__syncthreads();
			numer[u][k] = numer_sh[tid];
			denom[u][k] = denom_sh[tid];
		}
	}

}

//calculate matrix U and update prediction array
__device__ void calcualteU(int *tile_start, int thread_id, int tile_width, int start_u, int end_u, float *u_cu, float *v_cu, float *w_cu, float *wi_cu, int *row_num, int *row_value_index, float *row_value, float *numer, float *denom, float *prediction_items){
	int tid = threadIdx.x;
	//int tid_y = threadIdx.y;
	int index_i, length_start, length_end;
	int start_index;
	start_index = thread_id*tile_width;
	__shared__ float tmp_numer[64];
	for(int k=0; k<factors; k++){
		tmp_numer = 0;
		for(int u=tid+start_u; u<tid+end_u; u++){
			length_start = row_num[start_index];
			length_end = row_num[start_index+1];
			for(int i=length_start; i<length_end; i++){
				index_i = row_value_index[i];
				tmp_numer[tid] -= (w_cu[index_i]-wi_cu[index_i]) * prediction_items[u][index_i] * v_cu[index_i][k];
      }
      for(int k = 0; k<factors; k++){
        tmp_numer[tid] -= u_cu[u][k] * sv_cu[f][k];
      }
			u_cu[u][k] = (tmp_numer[tid]+numer[u][k])/(denom[u][k]+sv_cu[f][f]+reg);
			for(int i=length_start; i<length_end; i++){
				index_i = row_value_index[i];
				prediction_items[u][index_i] += u_cu[u][k] * v[index_i][k];
			}
		}
	}

}

void updateUserSchedule2(){
  int size, size2;

  //initial
  int userCount = 10;
  int itemCount = 20;
  int factors = 8;
  float reg = 0;
  size = 5;
  float trainMatrix_n[userCount];
  float W[userCount][size];
  float trainMatrix_spvin[userCount][size];
  float trainMatrix_spvdo[userCount][size];
  float U[userCount][factors];
  float V[itemCount][factors];
  float SV[factors][factors];
  for (int u = 0; u < userCount; u++){
    trainMatrix_n[u] = size;
    for(int i = 0; i < size; i++){
      W[u][i] = 0.5;
      trainMatrix_spvin[u][i] = i;
      trainMatrix_spvdo[u][i] = 1;
    }
    for(int i = 0; i < factors; i++){
      U[u][i] = (float)i;
    }
  }
  for (int u = 0; u < itemCount; u++){
    for(int i = 0; i < factors; i++)
      V[u][i] = 1;
  }
  for (int u = 0; u < factors; u++){
    for(int i = 0; i < factors; i++)
      SV[u][i] = 2;
  }

  float Wi[itemCount];
  for(int i=0; i<itemCount; i++){
    Wi[i] = ((float)i)/5;
  }

  
  float *v_col;
  //int total_size = trainMatrix.itemCount()+10;
  int total_size = 50+10;
  hipMalloc((void**)&v_col,sizeof(float)*total_size);

  float *w_cu, *u_cu, *v_cu, *sv_cu, *train_spvdo_cu;
  int *train_spvin_cu;
  hipMalloc((void**)&w_cu, sizeof(float)*total_size);
  hipMalloc((void**)&u_cu, sizeof(float)*userCount*factors);
  hipMalloc((void**)&v_cu, sizeof(float)*itemCount*factors);
  hipMalloc((void**)&sv_cu, sizeof(float)*factors*factors);
  hipMalloc((void**)&train_spvin_cu,sizeof(int)*total_size);
  hipMalloc((void**)&train_spvdo_cu,sizeof(float)*total_size);
  int *train_n, *train_n_cu;
  hipMalloc((void**)&train_n_cu,sizeof(int)*(userCount+1));
  float *w_h, *u_h, *v_h, *sv_h, *train_spvdo_h;
  int *train_spvin_h;
  hipHostAlloc((void**)&w_h, sizeof(float)*total_size, hipHostMallocDefault);
  hipHostAlloc((void**)&u_h, sizeof(float)*userCount*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&v_h, sizeof(float)*itemCount*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&sv_h, sizeof(float)*factors*factors, hipHostMallocDefault);
  hipHostAlloc((void**)&train_spvin_h, sizeof(int)*total_size, hipHostMallocDefault);
  hipHostAlloc((void**)&train_spvdo_h, sizeof(float)*total_size, hipHostMallocDefault);
  hipHostAlloc((void**)&train_n, sizeof(int)*(userCount+1), hipHostMallocDefault);
  
  train_n[0] = 0;
  for (int u = 0; u < userCount; u++){
    size = trainMatrix_n[u];
    size2 = train_n[u];
    train_n[u+1] = size2 + size;

    for(int i=0; i<size; i++){
      w_h[size2+i] = W[u][i];
      train_spvin_h[size2+i] = trainMatrix_spvin[u][i];
      train_spvdo_h[size2+i] = trainMatrix_spvdo[u][i];
    }
  }
  
  for (int u = 0; u < userCount; u++){
    for(int i=0; i<factors; i++)
      u_h[u*factors+i] = U[u][i];
  }
  
  for (int u = 0; u < itemCount; u++){
    for(int i=0; i<factors; i++)
      v_h[u*factors+i] = V[u][i];
  }
  for (int u = 0; u < factors; u++){
    for(int i=0; i<factors; i++)
      sv_h[u*factors+i] = SV[u][i];
  }
  hipMemcpy(u_cu, u_h, sizeof(float)*userCount*factors, hipMemcpyHostToDevice);
  hipMemcpy(v_cu, v_h, sizeof(float)*itemCount*factors, hipMemcpyHostToDevice);
  hipMemcpy(sv_cu, sv_h, sizeof(float)*factors*factors, hipMemcpyHostToDevice);
  hipMemcpy(w_cu, w_h, sizeof(float)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_spvdo_cu, train_spvdo_h, sizeof(float)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_spvin_cu, train_spvin_h, sizeof(int)*total_size, hipMemcpyHostToDevice);
  hipMemcpy(train_n_cu, train_n, sizeof(int)*(userCount+1), hipMemcpyHostToDevice);
  float *wi_cu;
  hipMalloc((void**)&wi_cu, sizeof(float)*itemCount);
  hipMemcpy(wi_cu, Wi, sizeof(float)*itemCount, hipMemcpyHostToDevice);

  float *u_numer, *u_denom;
  hipMalloc((void**)&u_numer,sizeof(float)*userCount*factors);
  hipMalloc((void**)&u_denom,sizeof(float)*userCount*factors);

  int dimx, dimy;
  dimx = 2;
  dimy = 2;
  //dim3 block(dimx, dimy);
  //dim3 grid(2);
  dim3 block(2);
  dim3 grid(dimx, dimy);
  updateUserCuda2<<<grid, block>>>(u_numer, u_denom, v_col, userCount, factors, reg, w_cu, u_cu, v_cu, sv_cu, wi_cu, train_spvdo_cu, train_n_cu, train_spvin_cu);
  //float u_numer_h[userCount*factors];
  //float u_denom_h[userCount*factors];
  //hipMemcpy(u_numer_h, u_numer, sizeof(float)*userCount*factors, hipMemcpyDeviceToHost);
  //hipMemcpy(u_denom_h, u_denom, sizeof(float)*userCount*factors, hipMemcpyDeviceToHost);
  hipMemcpy(u_h, u_cu, sizeof(float)*userCount*factors, hipMemcpyDeviceToHost);
  
  cout<<"cuda: \n";
  for (int u = 0; u < userCount; u++){
    for(int i=0; i<factors; i++){
      U[u][i] = u_h[u*factors+i];
      cout<<U[u][i]<<" ";
    }
  }
  cout<<endl;
  hipFree(v_col);
  hipFree(train_n_cu);
  hipFree(w_cu);
  hipFree(u_cu);
  hipFree(v_cu);
  hipFree(sv_cu);
  hipFree(train_spvin_cu);
  hipFree(train_spvdo_cu);
  hipFree(wi_cu);
  hipFree(u_numer);
  hipFree(u_denom);
  hipHostFree(train_n);
  hipHostFree(w_h);
  hipHostFree(u_h);
  hipHostFree(v_h);
  hipHostFree(sv_h);
  hipHostFree(train_spvin_h);
  hipHostFree(train_spvdo_h);
}

int main(){
  //updateUserSchedule1();
  //testUpdateSu();
  //testUpdateSv();
  //updateUserSchedule2();
  /*std::ifstream  fin;
	fin.open("amazon.rating");
	std::string line;
	int 
	if (!fin.is_open()) {
		fprintf(stderr, "Error: cannot open the file %s\n", dir.c_str());
		exit(EXIT_FAILURE);
	}

	float score;
	long timestamp = 0;
  int user_id, item_id, x = 0;
  


	while (std::getline(fin, line)) {
		std::istringstream word(line);
		word >> user_id;
		word >> item_id;
		word >> score;
		word >> timestamp;
		Rating rating(user_id,
			item_id,
			score,
			timestamp);
		if (user_ratings.size() < rating.userId + 1) {
			user_ratings.push_back(std::vector<Rating>());
		}
		user_ratings.at(rating.userId).push_back(rating);
		userCount = fmax(userCount, rating.userId);
		itemCount = fmax(itemCount, rating.itemId);
		x++;
	}

  */



  return 0;
}
